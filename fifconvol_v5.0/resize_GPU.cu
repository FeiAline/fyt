#include <cutil_inline.h>

//texture<float, 2, hipReadModeElementType> texref;

//////////////////////
//my area filter
//////////////////////

template<class T, class R>  // return type, texture type
__device__
R tex2D4N(const texture<T, 2, hipReadModeNormalizedFloat> tex_ref, float x, float y)
{
	R r;
	r = 0.25f * (tex2D(tex_ref, x-0.5f, y-0.5f)+tex2D(tex_ref, x-0.5f, y+0.5f)+
		     tex2D(tex_ref, x+0.5f, y-0.5f)+tex2D(tex_ref, x+0.5f, y+0.5f)); 

    return r;
}

template<class T, class R>  // return type, texture type
__device__
R tex2D16N(const texture<T, 2, hipReadModeNormalizedFloat> tex_ref, float x, float y)
{
	R r;
	r = 0.0625f * (tex2D(tex_ref, x-1.5f, y-1.5f) + tex2D(tex_ref, x-0.5f, y-1.5f) + tex2D(tex_ref, x+0.5f, y-1.5f) + tex2D(tex_ref, x+1.5f, y-1.5f)+
		       tex2D(tex_ref, x-1.5f, y-0.5f) + tex2D(tex_ref, x-0.5f, y-0.5f) + tex2D(tex_ref, x+0.5f, y-0.5f) + tex2D(tex_ref, x+1.5f, y-0.5f)+
		       tex2D(tex_ref, x-1.5f, y+0.5f) + tex2D(tex_ref, x-0.5f, y+0.5f) + tex2D(tex_ref, x+0.5f, y+0.5f) + tex2D(tex_ref, x+1.5f, y+0.5f)+
		       tex2D(tex_ref, x-1.5f, y+1.5f) + tex2D(tex_ref, x-0.5f, y+1.5f) + tex2D(tex_ref, x+0.5f, y+1.5f) + tex2D(tex_ref, x+1.5f, y+1.5f)); 
    return r;
}


/*__device__
float tex2D4N(const texture<unsigned char, 2, hipReadModeElementType> tex_ref, float x, float y)
{
	float r;
	r = 0.25f * (tex2D(tex_ref, x-0.5f, y-0.5f)+tex2D(tex_ref, x-0.5f, y+0.5f)+
		     tex2D(tex_ref, x+0.5f, y-0.5f)+tex2D(tex_ref, x+0.5f, y+0.5f)); 

    return r;
}

__device__
float tex2D16N(const texture<unsigned char, 2, hipReadModeElementType> tex_ref, float x, float y)
{
	float r;
	r = 0.0625f * (tex2D(tex_ref, x-1.5f, y-1.5f) + tex2D(tex_ref, x-0.5f, y-1.5f) + tex2D(tex_ref, x+0.5f, y-1.5f) + tex2D(tex_ref, x+1.5f, y-1.5f)+
		       tex2D(tex_ref, x-1.5f, y-0.5f) + tex2D(tex_ref, x-0.5f, y-0.5f) + tex2D(tex_ref, x+0.5f, y-0.5f) + tex2D(tex_ref, x+1.5f, y-0.5f)+
		       tex2D(tex_ref, x-1.5f, y+0.5f) + tex2D(tex_ref, x-0.5f, y+0.5f) + tex2D(tex_ref, x+0.5f, y+0.5f) + tex2D(tex_ref, x+1.5f, y+0.5f)+
		       tex2D(tex_ref, x-1.5f, y+1.5f) + tex2D(tex_ref, x-0.5f, y+1.5f) + tex2D(tex_ref, x+0.5f, y+1.5f) + tex2D(tex_ref, x+1.5f, y+1.5f)); 
    return r;
}*/

