#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cutil.h>
#include "resize_GPU.cu"
texture<float, 2, hipReadModeElementType> Texref;
texture<unsigned char, 2, hipReadModeNormalizedFloat> Texrefresz;
__constant__ float d_ker_h_r[25];
__constant__ float d_ker_h_i[25];
__constant__ float d_ker_d_r[25];
__constant__ float d_ker_d_i[25];
__constant__ float d_ker_v_r[25];
__constant__ float d_ker_v_i[25];
__constant__ float d_ker_ld_r[25];
__constant__ float d_ker_ld_i[25];

__global__ void Textureconvol_RP5_Arr_ori(short int* RP_out, int width)
{
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
    	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	const float x = (float)xIndex + 0.5;
	const float y = (float)yIndex + 0.5;

	//if (xIndex < width && yIndex < height)
	{
	float sum_h_r = 0.0f;
	float sum_h_i = 0.0f;
	float sum_d_r = 0.0f;
	float sum_d_i = 0.0f;
	float sum_v_r = 0.0f;
	float sum_v_i = 0.0f;
	float sum_ld_r= 0.0f;
	float sum_ld_i= 0.0f;
	float val;
	short int id;
	for (int i=-2; i<3; i++)
		for (int j=-2; j<3; j++)
		{
		val =255.0f*tex2D(Texrefresz, x+j, y+i);
		id  = (i+2)*5+j+2;
		sum_h_r += val*d_ker_h_r[id];
		sum_h_i += val*d_ker_h_i[id];
		sum_d_r += val*d_ker_d_r[id];
		sum_d_i += val*d_ker_d_i[id];
		sum_v_r += val*d_ker_v_r[id];
		sum_v_i += val*d_ker_v_i[id];
		sum_ld_r += val*d_ker_ld_r[id];
		sum_ld_i += val*d_ker_ld_i[id];
		}
	val = 0.0f;
	float ene_h = (sum_h_r*sum_h_r+sum_h_i*sum_h_i);
	val += ene_h;
	float ene_d = (sum_d_r*sum_d_r+sum_d_i*sum_d_i);
	val += ene_d;
	float ene_v = (sum_v_r*sum_v_r+sum_v_i*sum_v_i);
	val += ene_v;
	float ene_ld= (sum_ld_r*sum_ld_r+sum_ld_i*sum_ld_i);
	val += ene_ld;
	val = val/4.0f;
	//sum_h_r /= val; sum_h_i/=val;sum_d_r /=val; sum_d_i /=val;
	//sum_v_r /= val; sum_v_i/=val;sum_ld_r/=val; sum_ld_i/=val;
	id =0;
	id = 4*(!(ene_h>val));
	id += (ene_h>val)*(((sum_h_r+sum_h_i)>0.0f)*2 + ((sum_h_r-sum_h_i)>0.0f));

	id = id*5;
	id += 4*(!(ene_d>val));
	id += (ene_d>val)*(((sum_d_r+sum_d_i)>0.0f)*2 + ((sum_d_r-sum_d_i)>0.0f));


	id = id*5;
	id += 4*(!(ene_v>val));
	id +=(ene_v>val) *(((sum_v_r+sum_v_i)>0.0f)*2 + ((sum_v_r-sum_v_i)>0.0f));


	id = id*5;
	id += 4*(!(ene_ld>val));
	id += (ene_ld>val)*(((sum_ld_r+sum_ld_i)>0.0f)*2 + ((sum_ld_r-sum_ld_i)>0.0f));

	RP_out[yIndex*width+xIndex]=id;
	}
}

__global__ void Textureconvol_RP5_Arr(short int* RP_out, int width)
{
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
    	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	const float x = (float)xIndex + 0.5;
	const float y = (float)yIndex + 0.5;

	//if (xIndex < width && yIndex < height)
	{
	float sum_h_r = 0.0f;
	float sum_h_i = 0.0f;
	float sum_d_r = 0.0f;
	float sum_d_i = 0.0f;
	float sum_v_r = 0.0f;
	float sum_v_i = 0.0f;
	float sum_ld_r= 0.0f;
	float sum_ld_i= 0.0f;
	float val;
	short int id;
	for (int i=-2; i<3; i++)
		for (int j=-2; j<3; j++)
		{
		val = tex2D(Texref, x+j, y+i);
		id  = (i+2)*5+j+2;
		sum_h_r += val*d_ker_h_r[id];
		sum_h_i += val*d_ker_h_i[id];
		sum_d_r += val*d_ker_d_r[id];
		sum_d_i += val*d_ker_d_i[id];
		sum_v_r += val*d_ker_v_r[id];
		sum_v_i += val*d_ker_v_i[id];
		sum_ld_r += val*d_ker_ld_r[id];
		sum_ld_i += val*d_ker_ld_i[id];
		}
	val = 0.0f;
	float ene_h = (sum_h_r*sum_h_r+sum_h_i*sum_h_i);
	val += ene_h;
	float ene_d = (sum_d_r*sum_d_r+sum_d_i*sum_d_i);
	val += ene_d;
	float ene_v = (sum_v_r*sum_v_r+sum_v_i*sum_v_i);
	val += ene_v;
	float ene_ld= (sum_ld_r*sum_ld_r+sum_ld_i*sum_ld_i);
	val += ene_ld;
	val = val/4.0f;
	//sum_h_r /= val; sum_h_i/=val;sum_d_r /=val; sum_d_i /=val;
	//sum_v_r /= val; sum_v_i/=val;sum_ld_r/=val; sum_ld_i/=val;
	id =0;
	id = 4*(!(ene_h>val));
	id += (ene_h>val)*(((sum_h_r+sum_h_i)>0.0f)*2 + ((sum_h_r-sum_h_i)>0.0f));

	id = id*5;
	id += 4*(!(ene_d>val));
	id += (ene_d>val)*(((sum_d_r+sum_d_i)>0.0f)*2 + ((sum_d_r-sum_d_i)>0.0f));


	id = id*5;
	id += 4*(!(ene_v>val));
	id +=(ene_v>val) *(((sum_v_r+sum_v_i)>0.0f)*2 + ((sum_v_r-sum_v_i)>0.0f));


	id = id*5;
	id += 4*(!(ene_ld>val));
	id += (ene_ld>val)*(((sum_ld_r+sum_ld_i)>0.0f)*2 + ((sum_ld_r-sum_ld_i)>0.0f));

	RP_out[yIndex*width+xIndex]=id;
	}
}

__global__ void resize_bilinear(float* odata, float w_ratio, float h_ratio, int width)
{
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
    	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int Index = yIndex * width + xIndex;
	float index_row, index_col;
	//if (xIndex < (width) && yIndex < (height))
	{
		index_row = float(yIndex) * h_ratio;
		index_col = float(xIndex) * w_ratio;
		odata[Index] = 255.0f*tex2D(Texrefresz, index_col, index_row); 
	}
}

__global__ void resize_4N(float* odata, float w_ratio, float h_ratio, int width)
{
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
    	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int Index = yIndex * width + xIndex;
	float index_row, index_col;
	//if (xIndex < (width) && yIndex < (height))
	{
		index_row = float(yIndex) * h_ratio;
		index_col = float(xIndex) * w_ratio;
		odata[Index] = 255.0f*tex2D4N< unsigned char, float>(Texrefresz, index_col, index_row);//
	}
}

__global__ void resize_16N(float* odata, float w_ratio, float h_ratio, int width)
{
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
    	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int Index = yIndex * width + xIndex;
	float index_row, index_col;
	//if (xIndex < (width) && yIndex < (height))
	{
		index_row = float(yIndex) * h_ratio;
		index_col = float(xIndex) * w_ratio;
		odata[Index] = 255.0f*tex2D16N<unsigned char, float>(Texrefresz, index_col, index_row);//
	}
}

__global__ void Textureconvol_ene_Arr(float* ene_out, int width)
{
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
    	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	const float x = (float)xIndex + 0.5;
	const float y = (float)yIndex + 0.5;

	//if (xIndex < width && yIndex < height)
	{
	float sum_h_r = 0.0f;
	float sum_h_i = 0.0f;
	float sum_d_r = 0.0f;
	float sum_d_i = 0.0f;
	float sum_v_r = 0.0f;
	float sum_v_i = 0.0f;
	float sum_ld_r= 0.0f;
	float sum_ld_i= 0.0f;
	float val;
	int id;
	for (int i=-2; i<3; i++)
		for (int j=-2; j<3; j++)
		{
		val = tex2D(Texref, x+j, y+i);
		id  = (i+2)*5+j+2;
		sum_h_r += val*d_ker_h_r[id];
		sum_h_i += val*d_ker_h_i[id];
		sum_d_r += val*d_ker_d_r[id];
		sum_d_i += val*d_ker_d_i[id];
		sum_v_r += val*d_ker_v_r[id];
		sum_v_i += val*d_ker_v_i[id];
		sum_ld_r += val*d_ker_ld_r[id];
		sum_ld_i += val*d_ker_ld_i[id];
		}
	val = 0.0f;
	float ene_h = (sum_h_r*sum_h_r+sum_h_i*sum_h_i);
	val += ene_h;
	float ene_d = (sum_d_r*sum_d_r+sum_d_i*sum_d_i);
	val += ene_d;
	float ene_v = (sum_v_r*sum_v_r+sum_v_i*sum_v_i);
	val += ene_v;
	float ene_ld= (sum_ld_r*sum_ld_r+sum_ld_i*sum_ld_i);
	val += ene_ld;

	id=(yIndex*width+xIndex)*4;
	ene_out[id++]=sqrtf(ene_h/val);
	ene_out[id++]=sqrtf(ene_d/val);
	ene_out[id++]=sqrtf(ene_v/val);
	ene_out[id]=sqrtf(ene_ld/val);
	}
}

__global__ void Textureconvol_ene_Arr_thr(float* ene_out, float beta, int width)
{
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
    	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	const float x = (float)xIndex + 0.5;
	const float y = (float)yIndex + 0.5;

	//if (xIndex < width && yIndex < height)
	{
	float sum_h_r = 0.0f;
	float sum_h_i = 0.0f;
	float sum_d_r = 0.0f;
	float sum_d_i = 0.0f;
	float sum_v_r = 0.0f;
	float sum_v_i = 0.0f;
	float sum_ld_r= 0.0f;
	float sum_ld_i= 0.0f;
	float val;
	int id;
	for (int i=-2; i<3; i++)
		for (int j=-2; j<3; j++)
		{
		val = tex2D(Texref, x+j, y+i);
		id  = (i+2)*5+j+2;
		sum_h_r += val*d_ker_h_r[id];
		sum_h_i += val*d_ker_h_i[id];
		sum_d_r += val*d_ker_d_r[id];
		sum_d_i += val*d_ker_d_i[id];
		sum_v_r += val*d_ker_v_r[id];
		sum_v_i += val*d_ker_v_i[id];
		sum_ld_r += val*d_ker_ld_r[id];
		sum_ld_i += val*d_ker_ld_i[id];
		}
	val = 0.0f;
	float ene_h = (sum_h_r*sum_h_r+sum_h_i*sum_h_i);
	val += ene_h;
	float ene_d = (sum_d_r*sum_d_r+sum_d_i*sum_d_i);
	val += ene_d;
	float ene_v = (sum_v_r*sum_v_r+sum_v_i*sum_v_i);
	val += ene_v;
	float ene_ld= (sum_ld_r*sum_ld_r+sum_ld_i*sum_ld_i);
	val += ene_ld;
	val=max(val, beta*beta);

	id=(yIndex*width+xIndex)*4;
	ene_out[id++]=sqrtf(ene_h/val);
	ene_out[id++]=sqrtf(ene_d/val);
	ene_out[id++]=sqrtf(ene_v/val);
	ene_out[id]=sqrtf(ene_ld/val);
	}
}
