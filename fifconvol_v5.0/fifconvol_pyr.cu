#include "convol.cu"
#include "fifconvol_pyr.hpp"


fifconvol_pyr::fifconvol_pyr(unsigned int w, unsigned int h)
{
	d_w = w;
	d_h = h;
	p_w = iAlignUp(d_w, BLOCK_DIM);
	p_h = iAlignUp(d_h, BLOCK_HEI);
	U8Tex = hipCreateChannelDesc<unsigned char>();	
	F32Tex= hipCreateChannelDesc<float>();	
	hipMallocArray(&d_input, &U8Tex, p_w, p_h); 
	hipMallocArray(&d_inputf, &F32Tex, p_w, p_h); 
	CUDA_SAFE_CALL( hipMalloc( (void **)&d_resz, p_w*p_h*sizeof(float)));  
	CUDA_SAFE_CALL( hipMalloc( (void **)&d_RP, p_w*p_h*sizeof(short int)));
	CUDA_SAFE_CALL( hipMalloc( (void **)&d_ene, 4*p_w*p_h*sizeof(float)));  
	grid.x = iDivUp(d_w, BLOCK_DIM);
	grid.y = iDivUp(d_h, BLOCK_HEI);
	threads.x = BLOCK_DIM;
	threads.y = BLOCK_HEI;
	CUDA_SAFE_CALL( hipHostMalloc( (void **)&host_pad, p_w*p_h*sizeof(unsigned char)));  
	//setConvolutionKernels
	hipMemcpyToSymbol(HIP_SYMBOL(d_ker_h_r), ker_h_r, 25 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_ker_h_i), ker_h_i, 25 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_ker_d_r), ker_d_r, 25 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_ker_d_i), ker_d_i, 25 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_ker_v_r), ker_v_r, 25 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_ker_v_i), ker_v_i, 25 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_ker_ld_r), ker_ld_r, 25 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_ker_ld_i), ker_ld_i, 25 * sizeof(float));	
	Texref.filterMode = hipFilterModeLinear;	
	Texrefresz.filterMode = hipFilterModeLinear;	
}

fifconvol_pyr::~fifconvol_pyr()
{
	hipFreeArray(d_input);
	hipFreeArray(d_inputf);
	hipFree(d_resz);
	hipFree(d_RP);
	hipFree(host_pad);
	hipFree(d_ene);
}

void fifconvol_pyr::convol_RP5(const unsigned char* h_input, short int* h_RP)
{
	const unsigned char* host_in;
	host_in = h_input;
	if (d_w != p_w || d_h !=p_h)
	{
	pad_onhost(h_input);
	host_in = host_pad;
	}
	hipMemcpyToArray(d_input, 0, 0, host_in,p_w * p_h, hipMemcpyHostToDevice);
	// Set up the texture parameters for bilinear interpolation & clamping
	//hipUnbindTexture(Texref);
	hipUnbindTexture(Texrefresz);
	//hipBindTextureToArray(Texref, d_input);
	hipBindTextureToArray(Texrefresz, d_input);
	Textureconvol_RP5_Arr_ori<<<grid, threads>>>(d_RP, p_w);	
	CUDA_SAFE_CALL(hipMemcpy(h_RP, d_RP, p_w*p_h*sizeof(short int), hipMemcpyDeviceToHost));
}

void fifconvol_pyr::convol_RP5_d(short int* RP_out, int ori_width, int ori_height)
{
	float w_ratio=float(ori_width)/float(d_w);
	float h_ratio=float(ori_height)/float(d_h);
	if (w_ratio<2 && h_ratio<2)
		resize_bilinear<<<grid, threads>>>(d_resz, w_ratio, h_ratio, p_w);
	else if (w_ratio<4 && h_ratio<4)
		resize_4N<<<grid, threads>>>(d_resz, w_ratio, h_ratio, p_w);
	else
		resize_16N<<<grid, threads>>>(d_resz, w_ratio, h_ratio, p_w);
	hipMemcpyToArray(d_inputf, 0, 0, d_resz, p_w*p_h*4, hipMemcpyDeviceToDevice);
	hipUnbindTexture(Texref);
	hipBindTextureToArray(Texref, d_inputf);
	Textureconvol_RP5_Arr<<<grid, threads>>>(d_RP, p_w);	
	CUDA_SAFE_CALL(hipMemcpy(RP_out, d_RP, p_w*p_h*sizeof(short int), hipMemcpyDeviceToHost));
}

void fifconvol_pyr::convol_ene_d(float* ene_out, int ori_width, int ori_height)
{
	float w_ratio=float(ori_width)/float(d_w);
	float h_ratio=float(ori_height)/float(d_h);
	if (w_ratio<2 && h_ratio<2)
		resize_bilinear<<<grid, threads>>>(d_resz, w_ratio, h_ratio, p_w);
	else if (w_ratio<4 && h_ratio<4)
		resize_4N<<<grid, threads>>>(d_resz, w_ratio, h_ratio, p_w);
	else
		resize_16N<<<grid, threads>>>(d_resz, w_ratio, h_ratio, p_w);
	hipMemcpyToArray(d_inputf, 0, 0, d_resz, p_w*p_h*4, hipMemcpyDeviceToDevice);
	hipUnbindTexture(Texref);
	hipBindTextureToArray(Texref, d_inputf);
	Textureconvol_ene_Arr<<<grid, threads>>>(d_ene, p_w);
	CUDA_SAFE_CALL(hipMemcpy(ene_out, d_ene, 4*p_w*p_h*sizeof(float), hipMemcpyDeviceToHost));
}

void fifconvol_pyr::convol_ene_d_thr(float* ene_out, int ori_width, int ori_height, float beta)
{
	float w_ratio=float(ori_width)/float(d_w);
	float h_ratio=float(ori_height)/float(d_h);
	if (w_ratio<2 && h_ratio<2)
		resize_bilinear<<<grid, threads>>>(d_resz, w_ratio, h_ratio, p_w);
	else if (w_ratio<4 && h_ratio<4)
		resize_4N<<<grid, threads>>>(d_resz, w_ratio, h_ratio, p_w);
	else
		resize_16N<<<grid, threads>>>(d_resz, w_ratio, h_ratio, p_w);
	hipMemcpyToArray(d_inputf, 0, 0, d_resz, p_w*p_h*4, hipMemcpyDeviceToDevice);
	hipUnbindTexture(Texref);
	hipBindTextureToArray(Texref, d_inputf);
	Textureconvol_ene_Arr_thr<<<grid, threads>>>(d_ene, beta, p_w);
	CUDA_SAFE_CALL(hipMemcpy(ene_out, d_ene, 4*p_w*p_h*sizeof(float), hipMemcpyDeviceToHost));
}

void fifconvol_pyr::bind_input(const unsigned char* h_input)
{	
	const unsigned char* host_in = h_input;
	if (d_w != p_w || d_h !=p_h)
	{
	pad_onhost(h_input);
	host_in = host_pad;
	}
	hipMemcpyToArray(d_input, 0, 0, host_in,p_w * p_h, hipMemcpyHostToDevice);
	// Set up the texture parameters for bilinear interpolation & clamping
	hipUnbindTexture(Texrefresz);
	hipBindTextureToArray(Texrefresz, d_input);
}

void fifconvol_pyr::pad_onhost(const unsigned char* h_input)
{
	for (int i=0; i<p_h; i++)
		for (int j=0; j<p_w; j++)
		{
		if (i<d_h)
			{
			if (j<d_w)
			host_pad[i*p_w+j]=h_input[i*d_w+j];
			else
			host_pad[i*p_w+j]=h_input[i*d_w+d_w-1];
			}
		else
		host_pad[i*p_w+j]=host_pad[(d_h-1)*p_w+j];
		}
}

void fifconvol_pyr::get_padsz(int& w, int& h)
{
	w = p_w;
	h = p_h;
}

